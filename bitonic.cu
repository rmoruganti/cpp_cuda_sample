#include<stdio.h>
#include<ctime>
#include<cmath>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>



#define THREADS 1024 
__global__ void zeroPad(float *a, int arraySize, int SIZE)
{
	int blockId = blockIdx.x 
			 + blockIdx.y * gridDim.x 
			 + gridDim.x * gridDim.y * blockIdx.z; 
	int threadId = blockId * blockDim.x + threadIdx.x;

	
	if(threadId>=SIZE && threadId<arraySize)
	{
		a[threadId] = 0;
	}
}

__global__ void bitonic_sort_step(float *dev_values, int stage, int step, int newSize)

{
  unsigned int arrayIndex, xorVal; 
  int blockId = blockIdx.x 
			 + blockIdx.y * gridDim.x 
			 + gridDim.x * gridDim.y * blockIdx.z; 

  arrayIndex = blockId * blockDim.x + threadIdx.x;

  xorVal = arrayIndex^stage;

 
  //amongst the two threads that are being compared pick the lower thread and as each thread holds an array element 
  //pick the ones with threadID less than the size of the array being sorted 

  if ((xorVal)>arrayIndex && arrayIndex<newSize) {
	// sort the data in ascending order
    if ((arrayIndex&step)==0) {

	//swap if not in the right order
      if (dev_values[arrayIndex]>dev_values[xorVal]) {

        float temp = dev_values[arrayIndex];

        dev_values[arrayIndex] = dev_values[xorVal];

        dev_values[xorVal] = temp;

      }
    }
	// sort the data in descending order
    if ((arrayIndex&step)!=0) {

      //swap if not in the right order
      if (dev_values[arrayIndex]<dev_values[xorVal]) {

        float temp = dev_values[arrayIndex];

        dev_values[arrayIndex] = dev_values[xorVal];

        dev_values[xorVal] = temp;

      }
    }
  }

}
 


float bitonic_sort(float *values, int newSize, int SIZE)

{
  float *dev_values,tgpu = -1;
  clock_t start, end;
  hipError_t status;

 
  status = hipMalloc((void**) &dev_values, newSize * sizeof(float));
  
  status = hipMemcpy(dev_values, values, SIZE * sizeof(float), hipMemcpyHostToDevice);
    
  dim3 dimBlock(THREADS,1);  
  //decide on the grid size based on the array size
  int grid1 = ceil((float)newSize/(float)THREADS);

	int grid2 = 1;
	int grid3=1;
	if(grid1>65535)
	{
		grid2 =   ceil((float)newSize/(float)THREADS);
		grid1 = 65535;
	}
	if(grid2>65535)
	{
		grid3 = ceil((float)newSize/(float)THREADS);
		grid2 = 65535;
	}
	dim3 dimGrid(grid1,grid2,grid3); 
     



 
  int stage, step;

  start = clock();
  //pad extra zeros when the actual array size is not a power of 2
  if(newSize!=SIZE)
  {
  zeroPad<<<dimGrid, dimBlock>>>(dev_values,newSize,SIZE);
		status = hipDeviceSynchronize();
  }
  //This loop computes each step of the sorting network
  for (step = 2; step <= newSize; step <<= 1) {

  //This computes each stage in a step
    for (stage=step>>1; stage>0; stage=stage>>1) {

      bitonic_sort_step<<<dimGrid, dimBlock>>>(dev_values, stage, step, newSize);
	  status = hipDeviceSynchronize();
    }
  }
  end = clock();
  // Check for errors and exit on any failures
		status = hipGetLastError();
		if (status != hipSuccess) 
		{
		std::cout << "\n Kernel failed: " << hipGetErrorString(status) << 
		std::endl;
		
		}
 //if everything is successfully cpy back the results
		else{
			// if the actual array size is a power of 2 copy the entire array
		if(newSize==SIZE)
		hipMemcpy(values, dev_values, newSize * sizeof(float), hipMemcpyDeviceToHost);
		//else discard the zeros that are padded and copy the actual sorted entries of the array
		else
		hipMemcpy(values, dev_values+(newSize-SIZE), SIZE * sizeof(float), hipMemcpyDeviceToHost);
		tgpu = ((float)((end - start)*1000))/((float)(CLOCKS_PER_SEC));
		}
  hipFree(dev_values);
  // Returns the time for which the actual computation was being executed on the device 
  return tgpu;
}
 

